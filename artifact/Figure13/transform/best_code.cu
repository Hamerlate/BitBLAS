#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(128) Fused(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C) {
  
  half C_warp[512];
  __shared__ half A_shared[16384];
  __shared__ half B_shared[4096];
  half A_shared_warp[128];
  half B_shared_warp[32];

  const int MAX_BLOCK_N = 10;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int i_2_init = 0; i_2_init < 16; ++i_2_init) {
    for (int j_2_init = 0; j_2_init < 4; ++j_2_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[((i_2_init * 32) + (j_2_init * 8)) + i] = 0.0;}
;
    }
  }
  for (int k_0 = 0; k_0 < 256; ++k_0) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_ax2_ax3_0_fused_0 = 0; ax0_ax1_ax2_ax3_0_fused_0 < 16; ++ax0_ax1_ax2_ax3_0_fused_0) {
      *(uint4*)(A_shared + ((((ax0_ax1_ax2_ax3_0_fused_0 * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(A + ((((((((int)blockIdx.y) * 4194304) + (ax0_ax1_ax2_ax3_0_fused_0 * 262144)) + (((int)threadIdx.y) * 131072)) + (k_0 * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)));
    }
    #pragma unroll
    for (int ax0_ax1_ax2_ax3_0_fused_0_1 = 0; ax0_ax1_ax2_ax3_0_fused_0_1 < 4; ++ax0_ax1_ax2_ax3_0_fused_0_1) {
      *(uint4*)(B_shared + ((((ax0_ax1_ax2_ax3_0_fused_0_1 * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(B + ((((((((int)blockIdx.x) * 1048576) + (ax0_ax1_ax2_ax3_0_fused_0_1 * 262144)) + (((int)threadIdx.y) * 131072)) + (k_0 * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)));
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {
      for (int ax0 = 0; ax0 < 16; ++ax0) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_shared[(((((int)threadIdx.y) * 8192) + (ax0 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[(((((int)threadIdx.y) * 8192) + (ax0 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp + (ax0 * 8)))[0]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 8)))[1]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 8)))[2]), "=r"(((unsigned *)(A_shared_warp + (ax0 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax0_1 = 0; ax0_1 < 4; ++ax0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_shared[(((((int)threadIdx.z) * 2048) + (ax0_1 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_shared[(((((int)threadIdx.z) * 2048) + (ax0_1 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 8)))[0]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 8)))[1]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 8)))[2]), "=r"(((unsigned *)(B_shared_warp + (ax0_1 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int i_2 = 0; i_2 < 16; ++i_2) {
        for (int j_2 = 0; j_2 < 4; ++j_2) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + ((i_2 * 32) + (j_2 * 8))))[0]), "=r"(((unsigned *)(C_warp + ((i_2 * 32) + (j_2 * 8))))[1])
      : "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[0]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[1]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[2]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[3]), "r"(((unsigned *)(B_shared_warp + (j_2 * 8)))[0]), "r"(((unsigned *)(B_shared_warp + (j_2 * 8)))[1]), "r"(((unsigned *)(C_warp + ((i_2 * 32) + (j_2 * 8))))[0]), "r"(((unsigned *)(C_warp + ((i_2 * 32) + (j_2 * 8))))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + (((i_2 * 32) + (j_2 * 8)) + 4)))[0]), "=r"(((unsigned *)(C_warp + (((i_2 * 32) + (j_2 * 8)) + 4)))[1])
      : "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[0]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[1]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[2]), "r"(((unsigned *)(A_shared_warp + (i_2 * 8)))[3]), "r"(((unsigned *)(B_shared_warp + ((j_2 * 8) + 4)))[0]), "r"(((unsigned *)(B_shared_warp + ((j_2 * 8) + 4)))[1]), "r"(((unsigned *)(C_warp + (((i_2 * 32) + (j_2 * 8)) + 4)))[0]), "r"(((unsigned *)(C_warp + (((i_2 * 32) + (j_2 * 8)) + 4)))[1]));
  }
        }
      }
    }
  }
  for (int ax0_2 = 0; ax0_2 < 16; ++ax0_2) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      for (int local_id = 0; local_id < 8; local_id+=2) {
*((uint *)&(&(C[((((((((int)blockIdx.y) * 524288) + (((int)threadIdx.y) * 262144)) + (ax0_2 * 16384)) + (((int)blockIdx.x) * 2048)) + (((int)threadIdx.z) * 1024)) + (ax1 * 256))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))]) = *((uint *)&C_warp[((ax0_2 * 32) + (ax1 * 8)) + local_id]);
}
;
    }
  }
}

