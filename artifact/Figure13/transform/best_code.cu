#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(128) Fused(int8_t* __restrict__ A, int8_t* __restrict__ B, uint8_t* __restrict__ AScales, uint8_t* __restrict__ BScales, float* __restrict__ C) {
  
  float C_warp[16];
  signed char A_local[8];
  half A_decode_local[8];
  __shared__ half A_decode_shared[512];
  __shared__ half B_decode_shared[4096];
  signed char B_local[8];
  half B_decode_local[8];
  half A_decode_shared_warp[8];
  half B_decode_shared_warp[16];

  const int MAX_BLOCK_N = 10;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int i_2_init = 0; i_2_init < 1; ++i_2_init) {
    for (int j_2_init = 0; j_2_init < 2; ++j_2_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[(j_2_init * 8) + i] = 0.0;}
;
    }
  }
  for (int k_0 = 0; k_0 < 896; ++k_0) {
    if (((int)threadIdx.z) < 2) {
      *(int2*)(A_local + 0) = *(int2*)(A + ((((((int)blockIdx.y) * 458752) + (k_0 * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)));
    }
    for (int ax0 = 0; ax0 < 8; ++ax0) {
      if (((int)threadIdx.z) < 2) {
          uint __1 = ((max((((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) + ((uint)AScales[(((k_0 * 8192) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) >> 1))])), (uint)63) | ((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)7) << (uint)8)) << (uint)7) | (((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) & (uint)2);
        A_decode_local[ax0] = (*(half *)(&(__1)));
      }
    }
    __syncthreads();
    if (((int)threadIdx.z) < 2) {
      *(uint4*)(A_decode_shared + ((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 8))) = *(uint4*)(A_decode_local + 0);
    }
    for (int ax0_ax1_ax2_ax3_0_fused_0 = 0; ax0_ax1_ax2_ax3_0_fused_0 < 4; ++ax0_ax1_ax2_ax3_0_fused_0) {
      *(int2*)(B_local + 0) = *(int2*)(B + ((((((((int)blockIdx.x) * 3670016) + (ax0_ax1_ax2_ax3_0_fused_0 * 917504)) + ((((int)threadIdx.z) >> 1) * 458752)) + (k_0 * 512)) + ((((int)threadIdx.z) & 1) * 256)) + (((int)threadIdx.x) * 8)));
      for (int ax0_1 = 0; ax0_1 < 8; ++ax0_1) {
          uint __2 = ((max((((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) + ((uint)BScales[(((((k_0 * 8192) + (((int)blockIdx.x) * 128)) + (ax0_ax1_ax2_ax3_0_fused_0 * 32)) + ((((int)threadIdx.z) >> 1) * 16)) + (((int)threadIdx.x) >> 1))])), (uint)63) | ((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)7) << (uint)8)) << (uint)7) | (((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) & (uint)2);
        B_decode_local[ax0_1] = (*(half *)(&(__2)));
      }
      *(uint4*)(B_decode_shared + (((ax0_ax1_ax2_ax3_0_fused_0 * 1024) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(B_decode_local + 0);
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_decode_shared[(k_1 * 256)])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_decode_shared[(k_1 * 256)])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[3])
      : "r"(addr)
    );
  }
      for (int ax0_2 = 0; ax0_2 < 2; ++ax0_2) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 1024) + (ax0_2 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 1024) + (ax0_2 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_shared_warp + (ax0_2 * 8)))[0]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_2 * 8)))[1]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_2 * 8)))[2]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_2 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int j_2 = 0; j_2 < 2; ++j_2) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=f"(((float *)(C_warp + (j_2 * 8)))[0]), "=f"(((float *)(C_warp + (j_2 * 8)))[1]), "=f"(((float *)(C_warp + (j_2 * 8)))[2]), "=f"(((float *)(C_warp + (j_2 * 8)))[3])
      : "r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "r"(((unsigned *)(A_decode_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + (j_2 * 8)))[0]), "r"(((unsigned *)(B_decode_shared_warp + (j_2 * 8)))[1]), "f"(((float *)(C_warp + (j_2 * 8)))[0]), "f"(((float *)(C_warp + (j_2 * 8)))[1]), "f"(((float *)(C_warp + (j_2 * 8)))[2]), "f"(((float *)(C_warp + (j_2 * 8)))[3]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=f"(((float *)(C_warp + ((j_2 * 8) + 4)))[0]), "=f"(((float *)(C_warp + ((j_2 * 8) + 4)))[1]), "=f"(((float *)(C_warp + ((j_2 * 8) + 4)))[2]), "=f"(((float *)(C_warp + ((j_2 * 8) + 4)))[3])
      : "r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "r"(((unsigned *)(A_decode_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + ((j_2 * 8) + 4)))[0]), "r"(((unsigned *)(B_decode_shared_warp + ((j_2 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_2 * 8) + 4)))[0]), "f"(((float *)(C_warp + ((j_2 * 8) + 4)))[1]), "f"(((float *)(C_warp + ((j_2 * 8) + 4)))[2]), "f"(((float *)(C_warp + ((j_2 * 8) + 4)))[3]));
  }
      }
    }
  }
  for (int ax1 = 0; ax1 < 2; ++ax1) {
    for (int local_id = 0; local_id < 8; ++local_id) {
(&(C[((((((int)blockIdx.y) * 131072) + (((int)blockIdx.x) * 2048)) + (((int)threadIdx.z) * 512)) + (ax1 * 256))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))] = C_warp[(ax1 * 8) + local_id];
}
;
  }
}

