#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(128) Fused(int8_t* __restrict__ A, int8_t* __restrict__ B, uint8_t* __restrict__ AScales, uint8_t* __restrict__ BScales, float* __restrict__ C) {
  
  float C_warp[8];
  signed char A_local[8];
  half A_decode_local[8];
  __shared__ half A_decode_shared[512];
  __shared__ half B_decode_shared[2048];
  signed char B_local[8];
  half B_decode_local[8];
  half A_decode_shared_warp[8];
  half B_decode_shared_warp[8];

  const int MAX_BLOCK_N = 10;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int i_2_init = 0; i_2_init < 1; ++i_2_init) {
    for (int j_2_init = 0; j_2_init < 1; ++j_2_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[0 + i] = 0.0;}
;
    }
  }
  for (int k_0 = 0; k_0 < 256; ++k_0) {
    if (((int)threadIdx.z) < 2) {
      *(int2*)(A_local + 0) = *(int2*)(A + ((((((int)blockIdx.y) * 131072) + (k_0 * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)));
    }
    for (int ax0 = 0; ax0 < 8; ++ax0) {
      if (((int)threadIdx.z) < 2) {
          uint __1 = ((max((((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) + ((uint)AScales[(((k_0 * 1024) + (((int)blockIdx.y) * 16)) + (((int)threadIdx.x) >> 1))])), (uint)63) | ((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)7) << (uint)8)) << (uint)7) | (((((((uint)A_local[ax0]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) & (uint)2);
        A_decode_local[ax0] = (*(half *)(&(__1)));
      }
    }
    __syncthreads();
    if (((int)threadIdx.z) < 2) {
      *(uint4*)(A_decode_shared + ((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 8))) = *(uint4*)(A_decode_local + 0);
    }
    for (int ax0_ax1_ax2_ax3_0_fused_0 = 0; ax0_ax1_ax2_ax3_0_fused_0 < 2; ++ax0_ax1_ax2_ax3_0_fused_0) {
      *(int2*)(B_local + 0) = *(int2*)(B + ((((((((int)blockIdx.x) * 524288) + (ax0_ax1_ax2_ax3_0_fused_0 * 262144)) + ((((int)threadIdx.z) >> 1) * 131072)) + (k_0 * 512)) + ((((int)threadIdx.z) & 1) * 256)) + (((int)threadIdx.x) * 8)));
      for (int ax0_1 = 0; ax0_1 < 8; ++ax0_1) {
          uint __2 = ((max((((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) + ((uint)BScales[(((((k_0 * 1024) + (((int)blockIdx.x) * 64)) + (ax0_ax1_ax2_ax3_0_fused_0 * 32)) + ((((int)threadIdx.z) >> 1) * 16)) + (((int)threadIdx.x) >> 1))])), (uint)63) | ((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)7) << (uint)8)) << (uint)7) | (((((((uint)B_local[ax0_1]) >> (uint)0) & (uint)255) >> (uint)2) & (uint)31) & (uint)2);
        B_decode_local[ax0_1] = (*(half *)(&(__2)));
      }
      *(uint4*)(B_decode_shared + (((ax0_ax1_ax2_ax3_0_fused_0 * 1024) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(B_decode_local + 0);
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_decode_shared[(k_1 * 256)])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_decode_shared[(k_1 * 256)])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "=r"(((unsigned *)(A_decode_shared_warp + 0))[3])
      : "r"(addr)
    );
  }

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_shared[((((int)threadIdx.z) * 512) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_shared[((((int)threadIdx.z) * 512) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_shared_warp + 0))[0]), "=r"(((unsigned *)(B_decode_shared_warp + 0))[1]), "=r"(((unsigned *)(B_decode_shared_warp + 0))[2]), "=r"(((unsigned *)(B_decode_shared_warp + 0))[3])
      : "r"(addr)
    );
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=f"(((float *)(C_warp + 0))[0]), "=f"(((float *)(C_warp + 0))[1]), "=f"(((float *)(C_warp + 0))[2]), "=f"(((float *)(C_warp + 0))[3])
      : "r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "r"(((unsigned *)(A_decode_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + 0))[0]), "r"(((unsigned *)(B_decode_shared_warp + 0))[1]), "f"(((float *)(C_warp + 0))[0]), "f"(((float *)(C_warp + 0))[1]), "f"(((float *)(C_warp + 0))[2]), "f"(((float *)(C_warp + 0))[3]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
      :  "=f"(((float *)(C_warp + 4))[0]), "=f"(((float *)(C_warp + 4))[1]), "=f"(((float *)(C_warp + 4))[2]), "=f"(((float *)(C_warp + 4))[3])
      : "r"(((unsigned *)(A_decode_shared_warp + 0))[0]), "r"(((unsigned *)(A_decode_shared_warp + 0))[1]), "r"(((unsigned *)(A_decode_shared_warp + 0))[2]), "r"(((unsigned *)(A_decode_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + 4))[0]), "r"(((unsigned *)(B_decode_shared_warp + 4))[1]), "f"(((float *)(C_warp + 4))[0]), "f"(((float *)(C_warp + 4))[1]), "f"(((float *)(C_warp + 4))[2]), "f"(((float *)(C_warp + 4))[3]));
  }
    }
  }
  for (int local_id = 0; local_id < 8; ++local_id) {
(&(C[(((((int)blockIdx.y) * 16384) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.z) * 256))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))] = C_warp[0 + local_id];
}
;
}

