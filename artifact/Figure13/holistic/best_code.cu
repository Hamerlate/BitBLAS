#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(128) Fused(half* __restrict__ A, int8_t* __restrict__ B, half* __restrict__ C) {
  
  half C_warp[64];
  __shared__ half A_shared[2048];
  __shared__ signed char B_shared[8192];
  __shared__ half B_decode_shared[8192];
  signed char B_shared_local[4];
  half B_decode_local[8];
  half A_shared_warp[8];
  half B_decode_shared_warp[64];
  signed char B_shared_local_1[4];
  half B_decode_local_1[8];
  half A_shared_warp_1[8];
  half B_decode_shared_warp_1[64];

  const int MAX_BLOCK_N = 10;
  const auto baseBlockIdx = blockIdx.x + gridDim.x *blockIdx.y;
  const auto totalPanel = (gridDim.x * gridDim.y +MAX_BLOCK_N * gridDim.x - 1) / (MAX_BLOCK_N * gridDim.x);
  const auto totalBlock = gridDim.x * gridDim.y;
  const auto panelIdx = baseBlockIdx / (MAX_BLOCK_N *gridDim.x);
  const auto strideLd = panelIdx + 1 < totalPanel ?MAX_BLOCK_N : (totalBlock - panelIdx * (MAX_BLOCK_N *gridDim.x)) / gridDim.x;
  const auto bx = (panelIdx & 1) ? gridDim.x -(baseBlockIdx - panelIdx * MAX_BLOCK_N * gridDim.x) /strideLd - 1 : (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) / strideLd;
  const auto by = (baseBlockIdx - panelIdx * MAX_BLOCK_N *gridDim.x) % strideLd + panelIdx * MAX_BLOCK_N;
  const auto bz = blockIdx.z;
  const dim3 blockIdx(bx, by, bz);
  
  for (int i_2_init = 0; i_2_init < 1; ++i_2_init) {
    for (int j_2_init = 0; j_2_init < 8; ++j_2_init) {
      for (int i = 0; i < 8; ++i) {
C_warp[(j_2_init * 8) + i] = 0.0;}
;
    }
  }
  #pragma unroll
  for (int ax0_ax1_ax2_ax3_0_fused_0 = 0; ax0_ax1_ax2_ax3_0_fused_0 < 1; ++ax0_ax1_ax2_ax3_0_fused_0) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + (((((int)threadIdx.y) * 512) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((int)threadIdx.y) * 512) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.y) * 917504) + (((int)threadIdx.y) * 458752)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)))), "n"(16)
    );
  }
  }
  for (int ax0_ax1_ax2_ax3_fused_0_0_0_0 = 0; ax0_ax1_ax2_ax3_fused_0_0_0_0 < 2; ++ax0_ax1_ax2_ax3_fused_0_0_0_0) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(B_shared + ((((ax0_ax1_ax2_ax3_fused_0_0_0_0 * 2048) + (((int)threadIdx.z) * 1024)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((ax0_ax1_ax2_ax3_fused_0_0_0_0 * 2048) + (((int)threadIdx.z) * 1024)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(B + ((((((((int)blockIdx.x) * 3670016) + (ax0_ax1_ax2_ax3_fused_0_0_0_0 * 1835008)) + (((int)threadIdx.z) * 917504)) + (((int)threadIdx.y) * 458752)) + ((((int)threadIdx.x) >> 4) * 229376)) + ((((int)threadIdx.x) & 15) * 16)))), "n"(16)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0 = 0; k_0 < 895; ++k_0) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_ax2_ax3_0_fused_0_1 = 0; ax0_ax1_ax2_ax3_0_fused_0_1 < 1; ++ax0_ax1_ax2_ax3_0_fused_0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(A_shared + ((((((k_0 + 1) & 1) * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((((k_0 + 1) & 1) * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.y) * 917504) + (((int)threadIdx.y) * 458752)) + (k_0 * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8)) + 512))), "n"(16)
    );
  }
    }
    for (int ax0_ax1_ax2_ax3_fused_0_0_0_0_1 = 0; ax0_ax1_ax2_ax3_fused_0_0_0_0_1 < 2; ++ax0_ax1_ax2_ax3_fused_0_0_0_0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)(B_shared + (((((((k_0 + 1) & 1) * 4096) + (ax0_ax1_ax2_ax3_fused_0_0_0_0_1 * 2048)) + (((int)threadIdx.z) * 1024)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16)))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((((k_0 + 1) & 1) * 4096) + (ax0_ax1_ax2_ax3_fused_0_0_0_0_1 * 2048)) + (((int)threadIdx.z) * 1024)) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.x) * 16))))
    );
#endif
    __asm__ __volatile__(
      #if TVM_ENABLE_L2_PREFETCH
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;"
      #else
        "cp.async.cg.shared.global [%0], [%1], %2;"
      #endif
        :: "r"(addr), "l"((void*)(B + ((((((((((int)blockIdx.x) * 3670016) + (ax0_ax1_ax2_ax3_fused_0_0_0_0_1 * 1835008)) + (((int)threadIdx.z) * 917504)) + (((int)threadIdx.y) * 458752)) + ((((int)threadIdx.x) >> 4) * 229376)) + (k_0 * 256)) + ((((int)threadIdx.x) & 15) * 16)) + 256))), "n"(16)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int ax0_ax1_ax2_ax3_0_fused_0_2 = 0; ax0_ax1_ax2_ax3_0_fused_0_2 < 8; ++ax0_ax1_ax2_ax3_0_fused_0_2) {
      *(int*)(B_shared_local + 0) = *(int*)(B_shared + ((((((k_0 & 1) * 4096) + (ax0_ax1_ax2_ax3_0_fused_0_2 * 512)) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.x) * 4)));
      for (int ax0 = 0; ax0 < 8; ++ax0) {
        B_decode_local[ax0] = ((half)((B_shared_local[(ax0 >> 1)] >> ((signed char)((ax0 & 1) * 4))) & (signed char)15));
      }
      *(uint4*)(B_decode_shared + ((((ax0_ax1_ax2_ax3_0_fused_0_2 * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(B_decode_local + 0);
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_shared[((((k_0 & 1) * 1024) + (((int)threadIdx.y) * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[((((k_0 & 1) * 1024) + (((int)threadIdx.y) * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp + 0))[0]), "=r"(((unsigned *)(A_shared_warp + 0))[1]), "=r"(((unsigned *)(A_shared_warp + 0))[2]), "=r"(((unsigned *)(A_shared_warp + 0))[3])
      : "r"(addr)
    );
  }
      for (int ax0_1 = 0; ax0_1 < 8; ++ax0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 4096) + (ax0_1 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 4096) + (ax0_1 * 512)) + (k_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_shared_warp + (ax0_1 * 8)))[0]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_1 * 8)))[1]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_1 * 8)))[2]), "=r"(((unsigned *)(B_decode_shared_warp + (ax0_1 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int j_2 = 0; j_2 < 8; ++j_2) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + (j_2 * 8)))[0]), "=r"(((unsigned *)(C_warp + (j_2 * 8)))[1])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + (j_2 * 8)))[0]), "r"(((unsigned *)(B_decode_shared_warp + (j_2 * 8)))[1]), "r"(((unsigned *)(C_warp + (j_2 * 8)))[0]), "r"(((unsigned *)(C_warp + (j_2 * 8)))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + ((j_2 * 8) + 4)))[0]), "=r"(((unsigned *)(C_warp + ((j_2 * 8) + 4)))[1])
      : "r"(((unsigned *)(A_shared_warp + 0))[0]), "r"(((unsigned *)(A_shared_warp + 0))[1]), "r"(((unsigned *)(A_shared_warp + 0))[2]), "r"(((unsigned *)(A_shared_warp + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp + ((j_2 * 8) + 4)))[0]), "r"(((unsigned *)(B_decode_shared_warp + ((j_2 * 8) + 4)))[1]), "r"(((unsigned *)(C_warp + ((j_2 * 8) + 4)))[0]), "r"(((unsigned *)(C_warp + ((j_2 * 8) + 4)))[1]));
  }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax0_ax1_ax2_ax3_0_fused_0_3 = 0; ax0_ax1_ax2_ax3_0_fused_0_3 < 8; ++ax0_ax1_ax2_ax3_0_fused_0_3) {
    *(int*)(B_shared_local_1 + 0) = *(int*)(B_shared + (((((ax0_ax1_ax2_ax3_0_fused_0_3 * 512) + (((int)threadIdx.y) * 256)) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.x) * 4)) + 4096));
    for (int ax0_2 = 0; ax0_2 < 8; ++ax0_2) {
      B_decode_local_1[ax0_2] = ((half)((B_shared_local_1[(ax0_2 >> 1)] >> ((signed char)((ax0_2 & 1) * 4))) & (signed char)15));
    }
    *(uint4*)(B_decode_shared + ((((ax0_ax1_ax2_ax3_0_fused_0_3 * 1024) + (((int)threadIdx.y) * 512)) + (((int)threadIdx.z) * 256)) + (((int)threadIdx.x) * 8))) = *(uint4*)(B_decode_local_1 + 0);
  }
  __syncthreads();
  for (int k_1_1 = 0; k_1_1 < 2; ++k_1_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_shared[(((((int)threadIdx.y) * 512) + (k_1_1 * 256)) + 1024)])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_shared[(((((int)threadIdx.y) * 512) + (k_1_1 * 256)) + 1024)])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_shared_warp_1 + 0))[0]), "=r"(((unsigned *)(A_shared_warp_1 + 0))[1]), "=r"(((unsigned *)(A_shared_warp_1 + 0))[2]), "=r"(((unsigned *)(A_shared_warp_1 + 0))[3])
      : "r"(addr)
    );
  }
    for (int ax0_3 = 0; ax0_3 < 8; ++ax0_3) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 4096) + (ax0_3 * 512)) + (k_1_1 * 256))])) + (((int)threadIdx.x) * 8))));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_decode_shared[(((((int)threadIdx.z) * 4096) + (ax0_3 * 512)) + (k_1_1 * 256))])) + (((int)threadIdx.x) * 8)))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_decode_shared_warp_1 + (ax0_3 * 8)))[0]), "=r"(((unsigned *)(B_decode_shared_warp_1 + (ax0_3 * 8)))[1]), "=r"(((unsigned *)(B_decode_shared_warp_1 + (ax0_3 * 8)))[2]), "=r"(((unsigned *)(B_decode_shared_warp_1 + (ax0_3 * 8)))[3])
      : "r"(addr)
    );
  }
    }
    for (int j_2_1 = 0; j_2_1 < 8; ++j_2_1) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + (j_2_1 * 8)))[0]), "=r"(((unsigned *)(C_warp + (j_2_1 * 8)))[1])
      : "r"(((unsigned *)(A_shared_warp_1 + 0))[0]), "r"(((unsigned *)(A_shared_warp_1 + 0))[1]), "r"(((unsigned *)(A_shared_warp_1 + 0))[2]), "r"(((unsigned *)(A_shared_warp_1 + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp_1 + (j_2_1 * 8)))[0]), "r"(((unsigned *)(B_decode_shared_warp_1 + (j_2_1 * 8)))[1]), "r"(((unsigned *)(C_warp + (j_2_1 * 8)))[0]), "r"(((unsigned *)(C_warp + (j_2_1 * 8)))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_warp + ((j_2_1 * 8) + 4)))[0]), "=r"(((unsigned *)(C_warp + ((j_2_1 * 8) + 4)))[1])
      : "r"(((unsigned *)(A_shared_warp_1 + 0))[0]), "r"(((unsigned *)(A_shared_warp_1 + 0))[1]), "r"(((unsigned *)(A_shared_warp_1 + 0))[2]), "r"(((unsigned *)(A_shared_warp_1 + 0))[3]), "r"(((unsigned *)(B_decode_shared_warp_1 + ((j_2_1 * 8) + 4)))[0]), "r"(((unsigned *)(B_decode_shared_warp_1 + ((j_2_1 * 8) + 4)))[1]), "r"(((unsigned *)(C_warp + ((j_2_1 * 8) + 4)))[0]), "r"(((unsigned *)(C_warp + ((j_2_1 * 8) + 4)))[1]));
  }
    }
  }
  for (int ax1 = 0; ax1 < 8; ++ax1) {
    for (int local_id = 0; local_id < 8; local_id+=2) {
*((uint *)&(&(C[(((((((int)blockIdx.y) * 262144) + (((int)threadIdx.y) * 131072)) + (((int)blockIdx.x) * 4096)) + (((int)threadIdx.z) * 2048)) + (ax1 * 256))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))]) = *((uint *)&C_warp[(ax1 * 8) + local_id]);
}
;
  }
}

